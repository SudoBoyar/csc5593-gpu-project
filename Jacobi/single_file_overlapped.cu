#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <unistd.h>

using namespace std;

// Shorthand for formatting and printing usage options to stderr
#define fpe(msg) fprintf(stderr, "\t%s\n", msg);

// Shorthand for handling CUDA errors.
#define HANDLE_ERROR(err)  ( HandleError( err, __FILE__, __LINE__ ) )

/*****************
 * CUDA Utilites *
 *****************/

void HandleError(hipError_t err, const char *file, int line) {
    //
    // Handle and report on CUDA errors.
    //
    if (err != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);

        exit(EXIT_FAILURE);
    }
}

void checkCUDAError(const char *msg, bool exitOnError) {
    //
    // Check cuda error and print result if appropriate.
    //
    hipError_t err = hipGetLastError();

    if (hipSuccess != err) {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
        if (exitOnError) {
            exit(-1);
        }
    }
}

void cleanupCuda(void) {
    //
    // Clean up CUDA resources.
    //

    //
    // Explicitly cleans up all runtime-related resources associated with the
    // calling host thread.
    //
    HANDLE_ERROR(
        hipDeviceReset()
    );
}

/*********************
 * End CUDA Utilites *
 *********************/

struct Args {
    bool debug = false;
    bool sequential = false;
    bool blocked = false;
    bool overlapped = false;
    // Data attributes
    int size = 1024, dimensions = 2, alloc_size;
    int xSize = 1, ySize = 1, zSize = 1;
    int xBlockSize = 1, yBlockSize = 1, zBlockSize = 1, tBlockSize;
    // Run attributes
    int grid_size = 1, block_count = -1, thread_count = -1, iterations = 1000;
};


void usage(char *prog_name, string msg) {
    if (msg.size() > 0) {
        fputs(msg.c_str(), stderr);
    }

    fprintf(stderr, "%s\n", prog_name);
    fprintf(stderr, "Options are:\n");
    fpe("-n<size> Set data size (default: 1024)");
    fpe("-d<dims> Set number of data dimensions (1, 2, or 3) (default: 2)");
    fpe("-g<size> Set grid size");
    fpe("-b<num>  Set block count");
    fpe("-t<num>  Set thread count");
    fpe("-i<iter> Number of iterations to perform (default: 1000)");
    fpe("-x<size> X Dimension");
    fpe("-y<size> Y Dimension");
    fpe("-z<size> Z Dimension");
    fpe("-T<size> T Dimension");
    fpe("-S       Execute sequential, CPU version");
    fpe("-B       Execute blocked sequential, CPU version");
    fpe("-O       Execute sequential overlapped tiling, CPU version");
    fpe("-D       Print debug info");
    fpe("-h       Print usage info (this message)");
    exit(EXIT_FAILURE);
}

Args parse_arguments(int argc, char *argv[]) {
    Args args = Args();

    int opt;
    // Parse args
    while ((opt = getopt(argc, argv, "n:d:g:b:t:i:x:y:z:T:hSBOD")) != -1) {
        switch (opt) {
            case 'D':
                args.debug = true;
                break;
            case 'S':
                args.sequential = true;
                break;
            case 'B':
                args.blocked = true;
                break;
            case 'O':
                args.overlapped = true;
                break;
            case 'n':
                args.size = atoi(optarg);
                break;
            case 'd':
                args.dimensions = atoi(optarg);
                break;
            case 'g':
                args.grid_size = atoi(optarg);
                break;
            case 'b':
                args.block_count = atoi(optarg);
                break;
            case 't':
                args.thread_count = atoi(optarg);
                break;
            case 'i':
                args.iterations = atoi(optarg);
                break;
            case 'x':
                args.xBlockSize = atoi(optarg);
                break;
            case 'X':
                args.xSize = atoi(optarg);
                break;
            case 'y':
                args.yBlockSize = atoi(optarg);
                break;
            case 'Y':
                args.ySize = atoi(optarg);
                break;
            case 'z':
                args.zBlockSize = atoi(optarg);
                break;
            case 'Z':
                args.zSize = atoi(optarg);
                break;
            case 'T':
                args.tBlockSize = atoi(optarg);
                break;
            case 'h':
                usage(argv[0], "");
                break;
            default:
                usage(argv[0], "Unrecognized option\n");
        }
    }

    // check sizes
    if (args.size <= 0) {
        cout << "Data size must be larger than 0" << endl;
        exit(EXIT_FAILURE);
    }

    if (args.dimensions <= 0 || args.dimensions >= 4) {
        cerr << "Data must be 1, 2, or 3 dimensions" << endl;
        exit(EXIT_FAILURE);
    }

    // Calculations
    if (args.dimensions == 1) {
        args.alloc_size = args.size;
    } else if (args.dimensions == 2) {
        args.alloc_size = args.size * args.size;
    } else {
        args.alloc_size = args.size * args.size * args.size;
    }

    if (args.thread_count > 0) {
        args.block_count = args.alloc_size / args.thread_count;
    } else if (args.block_count > 0) {
        args.thread_count = args.alloc_size / args.block_count;
    } else {
        args.thread_count = 16;
        args.block_count = args.alloc_size / args.thread_count;
    }

    return args;
}

typedef struct {
    int dimensions;
    int height;
    int width;
    int depth;
    float *elements;
} Matrix;

Matrix initialize_matrix(int dimensions, int width, int height = 1, int depth = 1) {
    Matrix data;

    if (dimensions == 1 && width > 1) {
        data.width = width;
        data.height = 1;
        data.depth = 1;
        data.elements = (float *) malloc(width * sizeof(float));

        data.elements[0] = 1.0;
        data.elements[width - 1] = 1.0;
    } else if (dimensions == 2 && width > 1 && height > 1) {
        data.width = width;
        data.height = height;
        data.depth = 1;
        data.elements = (float *) malloc(width * height * sizeof(float));

        for (int y = 0; y < height; y += height - 1) {
            for (int x = 0; x < width; x++) {
                data.elements[y * width + x] = 1.0;
            }
        }

        for (int y = 0; y < height; y++) {
            for (int x = 0; x < width; x += width - 1) {
                data.elements[y * width + x] = 1.0;
            }
        }
    } else if (dimensions == 3 && width > 1 && height > 1 && depth > 1) {
        data.width = width;
        data.height = height;
        data.depth = depth;
        data.elements = (float *) malloc(width * height * depth * sizeof(float));

        for (int z = 0; z < depth; z++) {
            // X = 0 & N planes
            for (int y = 0; y < height; y++) {
                for (int x = 0; x < width; x += width - 1) {
                    data.elements[z * width * height + y * width + x] = 1.0;
                }
            }

            // Y = 0 & N planes
            for (int y = 0; y < height; y += height - 1) {
                for (int x = 0; x < width; x++) {
                    data.elements[z * width * height + y * width + x] = 1.0;
                }
            }
        }

        // Z = 0 & N planes
        for (int z = 0; z < depth; z += depth - 1) {
            for (int y = 0; y < height; y++) {
                for (int x = 0; x < width; x++) {
                    data.elements[z * width * height + y * width + x] = 1.0;
                }
            }
        }
    } else {
        fprintf(stderr, "Improper dimension or size.");
        exit(1);
    }

    return data;
}

/****************
 * CUDA KERNELS *
 ****************/

#define TILE_WIDTH 4
#define TILE_HEIGHT 2
#define TILE_DEPTH 2
#define TILE_AGE 2
#define PER_THREAD_X 2
#define PER_THREAD_Y 2
#define PER_THREAD_Z 2
#define BLOCK_DIM_X TILE_WIDTH/PER_THREAD_X
#define BLOCK_DIM_Y TILE_HEIGHT/PER_THREAD_Y
#define BLOCK_DIM_Z TILE_DEPTH/PER_THREAD_Z

#define BlockStartX(x) x * TILE_WIDTH;
#define BlockStartY(y) y * TILE_HEIGHT;
#define BlockStartZ(z) z * TILE_DEPTH;

#define BlockReadStartX(x) BlockStartX(x) - TILE_AGE;
#define BlockReadEndX(x) BlockStartX(x) + TILE_WIDTH + TILE_AGE;
#define BlockReadStartY(y) BlockStartY(y) - TILE_AGE;
#define BlockReadEndY(y) BlockStartY(y) + TILE_WIDTH + TILE_AGE;

__global__ void jacobi1d(Matrix data, Matrix result) {
    int threadCol = threadIdx.x;
    int blockCol = blockIdx.x;

    // Max number of things any given thread could be responsible for when working with ONE of the overlapped sections
    int perThreadOverlappedCount = TILE_AGE / TILE_WIDTH + 1;

    int globalX[PER_THREAD_X + perThreadOverlappedCount + perThreadOverlappedCount];
    int sharedX[PER_THREAD_X + perThreadOverlappedCount + perThreadOverlappedCount];

    // Shared and local data arrays
    __shared__ float shared[TILE_AGE][(TILE_AGE + TILE_WIDTH + TILE_AGE)];
    float local[PER_THREAD_X + perThreadOverlappedCount + perThreadOverlappedCount];

    // Some useful bits of info
    int globalBlockStart = blockCol * TILE_WIDTH;
    int globalBlockReadStart = max(0, globalBlockStart - TILE_AGE);
    int globalBlockReadEnd = min(data.width, globalBlockStart + TILE_WIDTH + TILE_AGE);

    int isFirstBlock = blockCol == 0;
    int isLastBlock = (blockCol + 1) * TILE_WIDTH >= data.width;

    /**
     * Global Memory:
     *
     *   Block 0   Block 1   Block 2   Block 3   Block 4
     * | _ _ _ _ | _ _ _ _ | _ _ _ _ | _ _ _ _ | _ _ _ _ |
     *
     * If we're block 2, we need:
     *
     *   Block 0   Block 1   Block 2   Block 3   Block 4
     * | _ _ _ _ | _ _ _ _ | _ _ _ _ | _ _ _ _ | _ _ _ _ |
     *                     |  this   |
     *
     * And for a tile age of AGE we also need:
     *
     *   Block 0   Block 1   Block 2   Block 3   Block 4
     * | _ _ _ _ | _ _ _ _ | _ _ _ _ | _ _ _ _ | _ _ _ _ |
     *              | this |         | this |
     *
     * So what we end up with is
     *
     *   Block 0   Block 1   Block 2   Block 3   Block 4
     * | _ _ _ _ | _ _ _ _ | _ _ _ _ | _ _ _ _ | _ _ _ _ |
     *              | AGE  |  TLSIZE | AGE  |
     *
     * TILE_AGE + TILE_SIZE + TILE_AGE
     */

    // Read the block data itself into shared memory
#pragma unroll
    for (int x = 0; x < PER_THREAD_X; x++) {
        // Offset shared index by TILE_AGE to allow for the overlapped data
        int sharedX = threadCol + BLOCK_DIM_X * x + TILE_AGE;
        // Remove TILE_AGE offset
        int globalX = globalBlockStart + sharedX - TILE_AGE;

        shared[0][sharedX] = data.elements[globalX];
    }

    // Read the adjacent/overlapped data sections into shared memory
#pragma unroll
    for (int x = 0; x < TILE_AGE; x += BLOCK_DIM_X) {
        // Left hand side data
        int sharedX = x + threadCol;
        int globalX = globalBlockStart - TILE_AGE + sharedX;
        if (globalX >= 0 && globalX < globalBlockStart) {
            shared[0][sharedX] = data.elements[globalX];
        }

        sharedX = TILE_AGE + TILE_WIDTH + threadCol + x;
        globalX = globalBlockStart + TILE_WIDTH + sharedX - TILE_AGE;
        if (globalX < data.width) {
            shared[0][sharedX] = data.elements[globalX];
        }
    }

    __syncthreads();

    /*
     * Calculate Values
     */
#pragma unroll
    for (int t = 1; t <= TILE_AGE; t++) {
        // First let's do the block itself, since that's nice and easy
#pragma unroll
        for (int x = 0; x < PER_THREAD_X; x++) {
            int globX = globalX[x + perThreadOverlappedCount];
            int sharX = sharedX[x + perThreadOverlappedCount];

            if (globX > 0 && globX < data.width - 1) {
                // Calculate new value
                shared[t][x] =
                    (
                        shared[t-1][sharX] +
                        shared[t-1][sharX - 1] +
                        shared[t-1][sharX + 1]
                    ) / 3;
            } else if (globX == 0 || globX == data.width - 1) {
                // On the edge
                shared[t][x] = shared[t-1][sharX];
            } else {
                // Beyond the edge, shouldn't ever hit this unless we messed something up
            }
        }

        // Now the left overlapped regions
#pragma unroll
        for (int x = 0; x < perThreadOverlappedCount; x++) {
            int globX = globalX[x];
            int sharX = sharedX[x];

            if (globX > 0 && globX < data.width - 1) {
                shared[t][sharX] = (shared[t-1][sharX - 1] + shared[t-1][sharX] + shared[t-1][sharX + 1]) / 3;
            } else {
                shared[t][sharX] = shared[t-1][sharX];
            }
        }

        // And the right overlapped regions
#pragma unroll
        for (int x = 0; x < perThreadOverlappedCount; x++) {
            int globX = globalX[PER_THREAD_X + perThreadOverlappedCount + x];
            int sharX = sharedX[PER_THREAD_X + perThreadOverlappedCount + x];

            if (globX > 0) {
                shared[t][sharX] = (shared[t-1][sharX - 1] + shared[t-1][sharX] + shared[t-1][sharX + 1]) / 3;
            } else {
                shared[t][sharX] = shared[t-1][sharX];
            }
        }
    }

    __syncthreads();

#pragma unroll
    for (int x = 0; x < PER_THREAD_X; x++) {
        int globX = globalX[x + perThreadOverlappedCount];
        int sharX = sharedX[x + perThreadOverlappedCount];

        result.elements[globX] = shared[TILE_AGE][sharX];
    }

    for (int x = 0; x < PER_THREAD_X; x++) {
        result.elements[globalIndex[x]] = shared[x];
    }
}

__global__ void jacobi2d(Matrix data, Matrix result) {
    int threadRow = threadIdx.y;
    int threadCol = threadIdx.x;
    int blockRow = blockIdx.y;
    int blockCol = blockIdx.x;

    // Indexes so we don't have to recompute them.
    int globalIndex[PER_THREAD_Y][PER_THREAD_X];
    int globalX[PER_THREAD_X];
    int globalY[PER_THREAD_Y];
    int sharedX[PER_THREAD_X];
    int sharedY[PER_THREAD_Y];

    // Shared and local data arrays
    __shared__ float shared[TILE_HEIGHT + 2][TILE_WIDTH + 2];
    float local[PER_THREAD_Y][PER_THREAD_X];

    /*
     * Calculate indexes into the global and shared arrays
     */

    // X shared and global
#pragma unroll
    for (int x = 0; x < PER_THREAD_X; x++) {
        sharedX[x] = threadCol + blockDim.x * x + 1;
        globalX[x] = blockCol * TILE_WIDTH + sharedX[x] - 1;
    }

    // Y shared and global
#pragma unroll
    for (int y = 0; y < PER_THREAD_Y; y++) {
        sharedY[y] = threadRow + blockDim.y * y + 1;
        globalY[y] = blockRow * TILE_HEIGHT + sharedY[y] - 1;
    }

    // Global absolute index
#pragma unroll
    for (int y = 0; y < PER_THREAD_Y; y++) {
#pragma unroll
        for (int x = 0; x < PER_THREAD_X; x++) {
            globalIndex[y][x] = globalX[x] + globalY[y] * data.width;
        }
    }

    /*
     * Copy into shared memory
     */
#pragma unroll
    for (int y = 0; y < PER_THREAD_Y; y++) {
#pragma unroll
        for (int x = 0; x < PER_THREAD_X; x++) {
            /*
             * We want to be doing block-contiguous reads, e.g. for 2x2 block dimension, 2 per thread for x and y
             * we want the read pattern to look like:
             *
             * 11|22
             * 11|22
             * -----
             * 33|44
             * 33|44
             *
             * Optimizing the width for reads is the responsibility of the calling code.
             */
            shared[sharedY[y]][sharedX[x]] = data.elements[globalIndex[y][x]];
        }
    }

    // Copy below-block dependencies into shared memory
    if (threadRow == 0 && blockRow > 0) {
#pragma unroll
        for (int x = 0; x < PER_THREAD_X; x++) {
            shared[0][sharedX[x]] = data.elements[globalIndex[0][x] - data.width];
        }
    }

    // Copy above-block dependencies into shared memory
    if (threadRow == blockDim.y - 1 && (blockRow + 1) * TILE_HEIGHT < data.height - 1) {
#pragma unroll
        for (int x = 0; x < PER_THREAD_X; x++) {
            shared[TILE_HEIGHT + 1][sharedX[x]] = data.elements[globalIndex[PER_THREAD_Y - 1][x] + data.width];
        }
    }

    // Copy left-of-block dependencies into shared memory
    if (threadCol == 0 && blockCol > 0) {
#pragma unroll
        for (int y = 0; y < PER_THREAD_Y; y++) {
            shared[sharedY[y]][0] = data.elements[globalIndex[y][0] - 1];
        }
    }

    // Copy right-of-block dependencies into shared memory
    if (threadCol == blockDim.x - 1 && (blockCol + 1) * TILE_WIDTH < data.width) {
#pragma unroll
        for (int y = 0; y < PER_THREAD_Y; y++) {
            shared[sharedY[y]][TILE_WIDTH + 1] = data.elements[globalIndex[y][PER_THREAD_X - 1] + 1];
        }
    }

    __syncthreads();

    /*
     * Calculate Values
     */
#pragma unroll
    for (int y = 0; y < PER_THREAD_Y; y++) {
        int globY = globalY[y];
        int sharY = sharedY[y];
#pragma unroll
        for (int x = 0; x < PER_THREAD_X; x++) {
            int globX = globalX[x];
            int sharX = sharedX[x];

            if (globX > 0 && globX < data.width - 1 && globY > 0 && globY < data.height - 1) {
                // Calculate new value
                local[y][x] =
                    (
                        shared[sharY][sharX - 1] +
                        shared[sharY][sharX] +
                        shared[sharY][sharX + 1] +
                        shared[sharY - 1][sharX] +
                        shared[sharY + 1][sharX]
                    ) * 0.2;
            } else if (globX == 0 || globX == data.width - 1 || globY == 0 || globY == data.height - 1) {
                // On the edge
                local[y][x] = shared[sharY][sharX];
            } else {
                // Beyond the edge, shouldn't ever hit this unless we messed something up
            }
        }
    }

    __syncthreads();

#pragma unroll
    for (int y = 0; y < PER_THREAD_Y; y++) {
#pragma unroll
        for (int x = 0; x < PER_THREAD_X; x++) {
            result.elements[globalIndex[y][x]] = local[y][x];
        }
    }
}

__global__ void jacobi3d(Matrix data, Matrix result) {
    int threadCol = threadIdx.x;
    int threadRow = threadIdx.y;
    int threadDep = threadIdx.z;
    int blockCol = blockIdx.x;
    int blockRow = blockIdx.y;
    int blockDep = blockIdx.z;

    // Indexes so we don't have to recompute them.
    int globalIndex[PER_THREAD_Z][PER_THREAD_Y][PER_THREAD_X];
    int globalX[PER_THREAD_X];
    int globalY[PER_THREAD_Y];
    int globalZ[PER_THREAD_Z];
    int sharedX[PER_THREAD_X];
    int sharedY[PER_THREAD_Y];
    int sharedZ[PER_THREAD_Z];

    // Shared and local data arrays
    __shared__ float shared[TILE_DEPTH + 2][TILE_HEIGHT + 2][TILE_WIDTH + 2];
    float local[PER_THREAD_Z][PER_THREAD_Y][PER_THREAD_X];

    /*
     * Calculate indexes into the global and shared arrays
     */

    // X shared and global
#pragma unroll
    for (int x = 0; x < PER_THREAD_X; x++) {
        sharedX[x] = threadCol + blockDim.x * x + 1;
        globalX[x] = blockCol * TILE_WIDTH + sharedX[x] - 1;
    }

    // Y shared and global
#pragma unroll
    for (int y = 0; y < PER_THREAD_Y; y++) {
        sharedY[y] = threadRow + blockDim.y * y + 1;
        globalY[y] = blockRow * TILE_HEIGHT + sharedY[y] - 1;
    }

    // Z shared and global
#pragma unroll
    for (int z = 0; z < PER_THREAD_Z; z++) {
        sharedZ[z] = threadDep + blockDim.z * z + 1;
        globalZ[z] = blockDep * TILE_DEPTH + sharedZ[z] - 1;
    }

    // Global absolute index
#pragma unroll
    for (int z = 0; z < PER_THREAD_Z; z++) {
        int zTemp = globalZ[z] * data.width * data.height;
#pragma unroll
        for (int y = 0; y < PER_THREAD_Y; y++) {
            int yTemp = globalY[y] * data.width;
#pragma unroll
            for (int x = 0; x < PER_THREAD_X; x++) {
                globalIndex[z][y][x] = globalX[x] + yTemp + zTemp;
            }
        }
    }

    /*
     * Copy into shared memory
     */
#pragma unroll
    for (int z = 0; z < PER_THREAD_Z; z++) {
#pragma unroll
        for (int y = 0; y < PER_THREAD_Y; y++) {
#pragma unroll
            for (int x = 0; x < PER_THREAD_X; x++) {
                shared[sharedZ[z]][sharedY[y]][sharedX[x]] = data.elements[globalIndex[z][y][x]];
            }
        }
    }

    // Copy below-block dependencies into shared memory
    if (threadRow == 0 && blockRow > 0) {
#pragma unroll
        for (int z = 0; z < PER_THREAD_Z; z++) {
#pragma unroll
            for (int x = 0; x < PER_THREAD_X; x++) {
                shared[sharedZ[z]][0][sharedX[x]] = data.elements[globalIndex[z][0][x] - data.width];
            }
        }
    }

    // Copy above-block dependencies into shared memory
    if (threadRow == blockDim.y - 1 && (blockRow + 1) * TILE_HEIGHT < data.height - 1) {
#pragma unroll
        for (int z = 0; z < PER_THREAD_Z; z++) {
#pragma unroll
            for (int x = 0; x < PER_THREAD_X; x++) {
                shared[sharedZ[z]][TILE_HEIGHT + 1][sharedX[x]] = data.elements[globalIndex[z][PER_THREAD_Y - 1][x] + data.width];
            }
        }
    }

    // Copy left-of-block dependencies into shared memory
    if (threadCol == 0 && blockCol > 0) {
#pragma unroll
        for (int z = 0; z < PER_THREAD_Z; z++) {
#pragma unroll
            for (int y = 0; y < PER_THREAD_Y; y++) {
                shared[sharedZ[z]][sharedY[y]][0] = data.elements[globalIndex[z][y][0] - 1];
            }
        }
    }

    // Copy right-of-block dependencies into shared memory
    if (threadCol == blockDim.x - 1 && (blockCol + 1) * TILE_WIDTH < data.width) {
#pragma unroll
        for (int z = 0; z < PER_THREAD_Z; z++) {
#pragma unroll
            for (int y = 0; y < PER_THREAD_Y; y++) {
                shared[sharedZ[z]][sharedY[y]][TILE_WIDTH + 1] = data.elements[globalIndex[z][y][PER_THREAD_X - 1] + 1];
            }
        }
    }

    // Copy in-front-of-block dependencies into shared memory
    if (threadDep == 0 && blockDep > 0) {
#pragma unroll
        for (int y = 0; y < PER_THREAD_Y; y++) {
#pragma unroll
            for (int x = 0; x < PER_THREAD_X; x++) {
                shared[0][sharedY[y]][sharedX[x]] = data.elements[globalIndex[0][y][x] - data.width * data.height];
            }
        }
    }

    // Copy behind-block dependencies into shared memory
    if (threadDep == blockDim.z - 1 && (blockDep + 1) * TILE_DEPTH < data.depth) {
#pragma unroll
        for (int y = 0; y < PER_THREAD_Y; y++) {
#pragma unroll
            for (int x = 0; x < PER_THREAD_X; x++) {
                shared[TILE_DEPTH + 1][sharedY[y]][sharedX[x]] = data.elements[globalIndex[PER_THREAD_Z - 1][y][x] + data.width * data.height];
            }
        }
    }

    __syncthreads();

    /*
     * Calculate Values
     */
    for (int z = 0; z < PER_THREAD_Z; z++) {
        int globZ = globalZ[z];
        int sharZ = sharedZ[z];
#pragma unroll
        for (int y = 0; y < PER_THREAD_Y; y++) {
            int globY = globalY[y];
            int sharY = sharedY[y];
#pragma unroll
            for (int x = 0; x < PER_THREAD_X; x++) {
                int globX = globalX[x];
                int sharX = sharedX[x];

                if (globX > 0 && globX < data.width - 1 && globY > 0 && globY < data.height - 1 && globZ > 0 && globZ < data.depth - 1) {
                    // Calculate new value
                    local[z][y][x] =
                        (
                            shared[sharZ][sharY][sharX] +
                            shared[sharZ][sharY][sharX - 1] +
                            shared[sharZ][sharY][sharX + 1] +
                            shared[sharZ][sharY - 1][sharX] +
                            shared[sharZ][sharY + 1][sharX] +
                            shared[sharZ - 1][sharY][sharX] +
                            shared[sharZ + 1][sharY][sharX]
                        ) / 7;
                } else if (globX == 0 || globX == data.width - 1 || globY == 0 || globY == data.height - 1 || globZ == 0 || globZ == data.depth - 1) {
                    // On the edge
                    local[z][y][x] = shared[sharZ][sharY][sharX];
                } else {
                    // Beyond the edge, shouldn't ever hit this unless we messed something up
                }
            }
        }
    }

    __syncthreads();

#pragma unroll
    for (int z = 0; z < PER_THREAD_Z; z++) {
#pragma unroll
        for (int y = 0; y < PER_THREAD_Y; y++) {
#pragma unroll
            for (int x = 0; x < PER_THREAD_X; x++) {
                result.elements[globalIndex[z][y][x]] = local[z][y][x];
            }
        }
    }
}

/********************
 * END CUDA KERNELS *
 ********************/

Matrix initialize_device(Matrix A, bool copyToDevice) {
    Matrix deviceA;

    deviceA.width = A.width;
    deviceA.height = A.height;
    deviceA.depth = A.depth;
    deviceA.dimensions = A.dimensions;

    size_t sizeA = A.width * A.height * A.depth * sizeof(float);

    HANDLE_ERROR(hipMalloc((void **) &deviceA.elements, sizeA));
    if (copyToDevice) {
        HANDLE_ERROR(hipMemcpy(deviceA.elements, A.elements, sizeA, hipMemcpyHostToDevice));
    }

    return deviceA;
}

void callKernel(Args args, Matrix A, Matrix B) {
    Matrix deviceA, deviceB;

    deviceA = initialize_device(A, true);
    deviceB = initialize_device(B, false);

    if (args.dimensions == 1) {
        dim3 blocks(max(args.size / (args.xBlockSize / PER_THREAD_X) , 1));
        dim3 threads(args.xBlockSize);

        for (int t = 0; t < args.iterations; t++) {
            jacobi1d<<<blocks, threads>>>(deviceA, deviceB);
//            checkCUDAError("jacobi1d", true);
            swap(deviceA, deviceB);
        }
    } else if (args.dimensions == 2) {
        dim3 blocks(max(args.size / TILE_WIDTH, 1), max(args.size / TILE_HEIGHT, 1));
        dim3 threads(TILE_WIDTH / PER_THREAD_X, TILE_HEIGHT / PER_THREAD_Y);
        for (int t = 0; t < args.iterations; t++) {
            jacobi2d<<<blocks, threads>>>(deviceA, deviceB);
//            checkCUDAError("jacobi2d", true);
            swap(deviceA, deviceB);
        }
    } else {
        dim3 blocks(max(args.size / TILE_WIDTH, 1), max(args.size / TILE_HEIGHT, 1), max(args.size / TILE_DEPTH, 1));
        dim3 threads(TILE_WIDTH / PER_THREAD_X, TILE_HEIGHT / PER_THREAD_Y, TILE_DEPTH / PER_THREAD_Z);
        for (int t = 0; t < args.iterations; t++) {
            jacobi3d<<<blocks, threads>>>(deviceA, deviceB);
//            checkCUDAError("jacobi3d", true);
            swap(deviceA, deviceB);
        }
    }

    HANDLE_ERROR(hipMemcpy(B.elements, deviceA.elements, A.width * A.height * A.depth * sizeof(float), hipMemcpyDeviceToHost));
}

// Data output
void print_data(float *data, int size, int dimensions) {
    if (size > 13) {
        cerr << "Data too big to print\n" << endl;
        return;
    }

    if (dimensions == 1) {
        for (int x = 0; x < size; x++) {
            printf("%.3f ", data[x]);
        }
    } else if (dimensions == 2) {
        for (int y = 0; y < size; y++) {
            for (int x = 0; x < size; x++) {
                printf("%.3f ", data[y * size + x]);
            }
            cout << endl;
        }
    } else if (dimensions == 3) {
        for (int z = 0; z < size; z++) {
            for (int y = 0; y < size; y++) {
                for (int x = 0; x < size; x++) {
                    printf("%.3f ", data[z * size * size + y * size + x]);
                }
                cout << endl;
            }
            cout << endl;
        }
    }
    cout << endl << endl;
}

// Main
int main(int argc, char *argv[]) {
    Args args = parse_arguments(argc, argv);
    Matrix A, B;
    A = initialize_matrix(args.dimensions, args.size, args.size, args.size);
    B = initialize_matrix(args.dimensions, args.size, args.size, args.size);

    atexit(cleanupCuda);

    //if (args.debug) { print_data(data, args.size, args.dimensions); }
    callKernel(args, A, B);
    if (args.debug) { print_data(B.elements, args.size, args.dimensions); }
}
